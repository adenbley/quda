#include "hip/hip_runtime.h"

#include "gauge_force_quda.h"


#define GF_SITE_MATRIX_LOAD_TEX 1

#if (GF_SITE_MATRIX_LOAD_TEX == 1)

#define LOAD_EVEN_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE_TEX(siteLink0TexSingle, dir, idx, var)
#define LOAD_ODD_MATRIX(dir, idx, var) 	LOAD_MATRIX_12_SINGLE_TEX(siteLink1TexSingle, dir, idx, var)
#else
#define LOAD_EVEN_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE(linkEven, dir, idx, var)
#define LOAD_ODD_MATRIX(dir, idx, var) LOAD_MATRIX_12_SINGLE(linkOdd, dir, idx, var)
#endif


#define LOAD_MATRIX LOAD_MATRIX_12_SINGLE
#define LOAD_ANTI_HERMITIAN LOAD_ANTI_HERMITIAN_SINGLE
#define WRITE_ANTI_HERMITIAN WRITE_ANTI_HERMITIAN_SINGLE
#define RECONSTRUCT_MATRIX RECONSTRUCT_MATRIX_12_SINGLE


__constant__ int path_max_length;

void
gauge_force_init_cuda(QudaGaugeParam* param, int path_max_length)
{    
    static int gauge_force_init_cuda_flag = 0;
    if (gauge_force_init_cuda_flag){
	return;
    }
    gauge_force_init_cuda_flag=1;

    init_kernel_cuda(param);
    
    hipMemcpyToSymbol(HIP_SYMBOL("path_max_length"), &path_max_length, sizeof(int));

}

#define COMPUTE_NEW_FULL_IDX_PLUS_UPDATE(mydir, idx) do {		\
        switch(mydir){                                                  \
        case 0:                                                         \
            new_mem_idx = ( (new_x1==X1m1)?idx-X1m1:idx+1);		\
	    new_x1 = (new_x1==X1m1)?0:new_x1+1;				\
            break;                                                      \
        case 1:                                                         \
            new_mem_idx = ( (new_x2==X2m1)?idx-X2X1mX1:idx+X1);		\
	    new_x2 = (new_x2==X2m1)?0:new_x2+1;				\
            break;                                                      \
        case 2:                                                         \
            new_mem_idx = ( (new_x3==X3m1)?idx-X3X2X1mX2X1:idx+X2X1);	\
	    new_x3 = (new_x3==X3m1)?0:new_x3+1;				\
            break;                                                      \
        case 3:                                                         \
            new_mem_idx = ( (new_x4==X4m1)?idx-X4X3X2X1mX3X2X1:idx+X3X2X1); \
	    new_x4 = (new_x4==X4m1)?0:new_x4+1;				\
            break;                                                      \
        }                                                               \
    }while(0)

#define COMPUTE_NEW_FULL_IDX_MINUS_UPDATE(mydir, idx) do {		\
        switch(mydir){                                                  \
        case 0:                                                         \
            new_mem_idx = ( (new_x1==0)?idx+X1m1:idx-1);		\
	    new_x1 = (new_x1==0)?X1m1:new_x1 - 1;			\
            break;                                                      \
        case 1:                                                         \
            new_mem_idx = ( (new_x2==0)?idx+X2X1mX1:idx-X1);		\
	    new_x2 = (new_x2==0)?X2m1:new_x2 - 1;			\
            break;                                                      \
        case 2:                                                         \
            new_mem_idx = ( (new_x3==0)?idx+X3X2X1mX2X1:idx-X2X1);	\
	    new_x3 = (new_x3==0)?X3m1:new_x3 - 1;			\
            break;                                                      \
        case 3:                                                         \
            new_mem_idx = ( (new_x4==0)?idx+X4X3X2X1mX3X2X1:idx-X3X2X1); \
	    new_x4 = (new_x4==0)?X4m1:new_x4 - 1;			\
            break;                                                      \
        }                                                               \
    }while(0)

#define GF_COMPUTE_RECONSTRUCT_SIGN(sign, dir, i1,i2,i3,i4) do {	\
        sign =1;							\
        switch(dir){							\
        case XUP:							\
            if ( (i4 & 1) == 1){					\
                sign = 1;						\
            }								\
            break;							\
        case YUP:							\
            if ( ((i4+i1) & 1) == 1){					\
                sign = 1;						\
            }								\
            break;							\
        case ZUP:							\
            if ( ((i4+i1+i2) & 1) == 1){				\
                sign = 1;						\
            }								\
            break;							\
        case TUP:							\
            if (i4 == X4m1 ){						\
                sign = 1;						\
            }								\
            break;							\
        }								\
    }while (0)



//for now we only consider 12-reconstruct and single precision

template<int oddBit>
__global__ void
parity_compute_gauge_force_kernel(float2* momEven, float2* momOdd,
				  int dir, double eb3,
				  float4* linkEven, float4* linkOdd,
				  int* input_path, 
				  int* length, float* path_coeff, int num_paths)
{
    int i,j=0;
    int sid = blockIdx.x * blockDim.x + threadIdx.x;
    
    int z1 = FAST_INT_DIVIDE(sid, X1h);
    int x1h = sid - z1*X1h;
    int z2 = FAST_INT_DIVIDE(z1, X2);
    int x2 = z1 - z2*X2;
    int x4 = FAST_INT_DIVIDE(z2, X3);
    int x3 = z2 - x4*X3;
    int x1odd = (x2 + x3 + x4 + oddBit) & 1;
    int x1 = 2*x1h + x1odd;  
    int X = 2*sid + x1odd;
    
    int sign = 1;
    
    float2* mymom=momEven;
    if (oddBit){
	mymom = momOdd;
    }

    float4 LINKA0, LINKA1, LINKA2, LINKA3, LINKA4;
    float4 LINKB0, LINKB1, LINKB2, LINKB3, LINKB4;
    float2 STAPLE0, STAPLE1, STAPLE2, STAPLE3,STAPLE4, STAPLE5, STAPLE6, STAPLE7, STAPLE8;
    float2 AH0, AH1, AH2, AH3, AH4;

    int new_mem_idx;
    
    
    SET_SU3_MATRIX(staple, 0);
    for(i=0;i < num_paths; i++){
	int nbr_oddbit = (oddBit^1 );
	
	int new_x1 =x1;
	int new_x2 =x2;
	int new_x3 =x3;
	int new_x4 =x4;
	COMPUTE_NEW_FULL_IDX_PLUS_UPDATE(dir, X);
	
	//linka: current matrix
	//linkb: the loaded matrix in this round	
	SET_UNIT_SU3_MATRIX(linka);	
	int* path = input_path + i*path_max_length;
	
	int lnkdir;
	int path0 = path[0];
	if (GOES_FORWARDS(path0)){
	    lnkdir=path0;
	}else{
	    lnkdir=OPP_DIR(path0);
	    COMPUTE_NEW_FULL_IDX_MINUS_UPDATE(OPP_DIR(path0), new_mem_idx);
	    nbr_oddbit = nbr_oddbit^1;
	    
	}
	
	int nbr_idx = new_mem_idx >>1;
	if (nbr_oddbit){
	    LOAD_ODD_MATRIX( lnkdir, nbr_idx, LINKB);
	}else{
	    LOAD_EVEN_MATRIX( lnkdir, nbr_idx, LINKB);
	}
	
	GF_COMPUTE_RECONSTRUCT_SIGN(sign, lnkdir, new_x1, new_x2, new_x3, new_x4);
	RECONSTRUCT_MATRIX(lnkdir, nbr_idx, sign, linkb);
	if (GOES_FORWARDS(path0)){
	    COPY_SU3_MATRIX(linkb, linka);
	    COMPUTE_NEW_FULL_IDX_PLUS_UPDATE(path0, new_mem_idx);
	    nbr_oddbit = nbr_oddbit^1;
	}else{
	    SU3_ADJOINT(linkb, linka);
	}	
	
	for(j=1; j < length[i]; j++){
	    
	    int lnkdir;
	    int pathj = path[j];
	    if (GOES_FORWARDS(pathj)){
		lnkdir=pathj;
	    }else{
		lnkdir=OPP_DIR(pathj);
		COMPUTE_NEW_FULL_IDX_MINUS_UPDATE(OPP_DIR(pathj), new_mem_idx);
		nbr_oddbit = nbr_oddbit^1;

	    }
	    
	    int nbr_idx = new_mem_idx >>1;
	    if (nbr_oddbit){
		LOAD_ODD_MATRIX(lnkdir, nbr_idx, LINKB);
	    }else{
		LOAD_EVEN_MATRIX(lnkdir, nbr_idx, LINKB);
	    }
	    GF_COMPUTE_RECONSTRUCT_SIGN(sign, lnkdir, new_x1, new_x2, new_x3, new_x4);
	    RECONSTRUCT_MATRIX(lnkdir, nbr_idx, sign, linkb);
	    if (GOES_FORWARDS(pathj)){
		MULT_SU3_NN_TEST(linka, linkb);
		
		COMPUTE_NEW_FULL_IDX_PLUS_UPDATE(pathj, new_mem_idx);
		nbr_oddbit = nbr_oddbit^1;
		
		
	    }else{
		MULT_SU3_NA_TEST(linka, linkb);		
	    }
	    
	}//j
	SCALAR_MULT_ADD_SU3_MATRIX(staple, linka, path_coeff[i], staple);
    }//i
    

    //update mom 
    if (oddBit){
	LOAD_ODD_MATRIX(dir, sid, LINKA);
    }else{
	LOAD_EVEN_MATRIX(dir, sid, LINKA);
    }
    GF_COMPUTE_RECONSTRUCT_SIGN(sign, dir, x1, x2, x3, x4);
    RECONSTRUCT_MATRIX(dir, sid, sign, linka);
    MULT_SU3_NN_TEST(linka, staple);
    LOAD_ANTI_HERMITIAN(mymom, dir, sid, AH);
    UNCOMPRESS_ANTI_HERMITIAN(ah, linkb);
    SCALAR_MULT_SUB_SU3_MATRIX(linkb, linka, eb3, linka);
    MAKE_ANTI_HERMITIAN(linka, ah);
    
    WRITE_ANTI_HERMITIAN(mymom, dir, sid, AH);

    return;
}

void
gauge_force_cuda(FullMom  cudaMom, int dir, double eb3, FullGauge cudaSiteLink,
                 QudaGaugeParam* param, int** input_path, 
		 int* length, void* path_coeff, int num_paths, int max_length)
{

    int i, j;
    //input_path
    int bytes = num_paths*max_length* sizeof(int);
    int* input_path_d;
    hipMalloc((void**)&input_path_d, bytes); CUERR;    
    hipMemset(input_path_d, 0, bytes);CUERR;

    int* input_path_h = (int*)malloc(bytes);
    if (input_path_h == NULL){
	printf("ERROR: malloc failed for input_path_h in function %s\n", __FUNCTION__);
	exit(1);
    }
        
    memset(input_path_h, 0, bytes);
    for(i=0;i < num_paths;i++){
	for(j=0; j < length[i]; j++){
	    input_path_h[i*max_length + j] =input_path[i][j];
	}
    }

    hipMemcpy(input_path_d, input_path_h, bytes, hipMemcpyHostToDevice); CUERR;
    
    //length
    int* length_d;
    hipMalloc((void**)&length_d, num_paths*sizeof(int)); CUERR;
    hipMemcpy(length_d, length, num_paths*sizeof(int), hipMemcpyHostToDevice); CUERR;
    
    //path_coeff
    int gsize;
    if (param->cuda_prec == QUDA_DOUBLE_PRECISION){
	gsize = sizeof(double);
    }else{
	gsize= sizeof(float);
    }     
    void* path_coeff_d;
    hipMalloc((void**)&path_coeff_d, num_paths*gsize); CUERR;
    hipMemcpy(path_coeff_d, path_coeff, num_paths*gsize, hipMemcpyHostToDevice); CUERR;

    //compute the gauge forces
    int volume = param->X[0]*param->X[1]*param->X[2]*param->X[3];
    dim3 blockDim(cudaSiteLink.blockDim, 1,1);
    dim3 gridDim(volume/blockDim.x, 1, 1);
    dim3 halfGridDim(volume/(2*blockDim.x), 1, 1);
    
    float2* momEven = (float2*)cudaMom.even;
    float2* momOdd = (float2*)cudaMom.odd;
    float4* linkEven = (float4*)cudaSiteLink.even;
    float4* linkOdd = (float4*)cudaSiteLink.odd;        

    hipBindTexture(0, siteLink0TexSingle, cudaSiteLink.even, cudaSiteLink.bytes);
    hipBindTexture(0, siteLink1TexSingle, cudaSiteLink.odd, cudaSiteLink.bytes);
    parity_compute_gauge_force_kernel<0><<<halfGridDim, blockDim>>>(momEven, momOdd,
								  dir, eb3,
								  linkEven, linkOdd, 
								  input_path_d, length_d, (float*)path_coeff_d,
								  num_paths);   
    //odd
    /* The reason we do not switch the even/odd function input paramemters and the texture binding
     * is that we use the oddbit to decided where to load, in the kernel function
     */
    parity_compute_gauge_force_kernel<1><<<halfGridDim, blockDim>>>(momEven, momOdd,
								  dir, eb3,
								  linkEven, linkOdd, 
								  input_path_d, length_d, (float*)path_coeff_d,
								  num_paths);  
    

    
    hipUnbindTexture(siteLink0TexSingle);
    hipUnbindTexture(siteLink1TexSingle);
    
    CUERR;
    
    hipFree(input_path_d); CUERR;
    free(input_path_h);
    hipFree(length_d);
    hipFree(path_coeff_d);

    

}


#undef LOAD_EVEN_MATRIX
#undef LOAD_ODD_MATRIX
#undef LOAD_MATRIX 
#undef LOAD_ANTI_HERMITIAN 
#undef WRITE_ANTI_HERMITIAN
#undef RECONSTRUCT_MATRIX
