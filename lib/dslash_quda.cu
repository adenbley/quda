#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>

//these are access control for staggered action
#if (__CUDA_ARCH__ >= 200)
//#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
#define DIRECT_ACCESS_SPINOR
#else
#define DIRECT_ACCESS_FAT_LINK
//#define DIRECT_ACCESS_LONG_LINK
//#define DIRECT_ACCESS_SPINOR
#endif

#include <quda_internal.h>
#include <dslash_quda.h>

#define BLOCK_DIM 64

#include <dslash_textures.h>
#include <dslash_constants.h>

// kludge to avoid '#include nested too deeply' error
#define DD_DAG 0
#include <dslash_def.h>
#undef DD_DAG
#define DD_DAG 1
#include <dslash_def.h>
#undef DD_DAG

#include <clover_def.h> // kernels for applying the clover term alone

#include <dslash_staggered_def.h> // kernels for staggered kernels

#include <blas_quda.h>
#include <sys/time.h>
#include "mpicomm.h"
#include "exchange_face.h"

extern void *fwd_nbr_spinor, *back_nbr_spinor;
extern void* f_norm, *b_norm;
hipStream_t stream[2];

int dslashCudaSharedBytes(QudaPrecision precision) {
  return BLOCK_DIM*SHARED_FLOATS_PER_THREAD*precision;
}

template <int spinorN, typename spinorFloat, typename gaugeFloat>
void dslashCuda(spinorFloat *out, float *outNorm, const gaugeFloat *gauge0, const gaugeFloat *gauge1, 
		const QudaReconstructType reconstruct, const spinorFloat *in, const float *inNorm,
		const int parity, const int dagger, const spinorFloat *x, const float *xNorm, 
		const double &a, const int volume, const int length) {

  dim3 gridDim(volume/BLOCK_DIM, 1, 1);
  dim3 blockDim(BLOCK_DIM, 1, 1);

  int shared_bytes = blockDim.x*SHARED_FLOATS_PER_THREAD*bindSpinorTex<spinorN>(length, in, inNorm, x, xNorm);

  if (x==0) { // not doing xpay
    if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	dslash12Kernel <<<gridDim, blockDim, shared_bytes>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, parity);
      } else {
	dslash12DaggerKernel <<<gridDim, blockDim, shared_bytes>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, parity);
      }
    } else {
      if (!dagger) {
	dslash8Kernel <<<gridDim, blockDim, shared_bytes>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, parity);
      } else {
	dslash8DaggerKernel <<<gridDim, blockDim, shared_bytes>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, parity);
      }
    }
  } else { // doing xpay
    if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	dslash12XpayKernel <<<gridDim, blockDim, shared_bytes>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, parity, x, xNorm, a);
      } else {
	dslash12DaggerXpayKernel <<<gridDim, blockDim, shared_bytes>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, parity, x, xNorm, a);
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_8) {
      if (!dagger) {
	dslash8XpayKernel <<<gridDim, blockDim, shared_bytes>>> 
	  (out, outNorm, gauge0, gauge1, in, inNorm, parity, x, xNorm, a);
      } else {
	dslash8DaggerXpayKernel <<<gridDim, blockDim, shared_bytes>>>
	  (out, outNorm, gauge0, gauge1, in, inNorm, parity, x, xNorm, a);
      }
    }
  }
  
}

// Wilson wrappers
void dslashCuda(void *out, void *outNorm, const FullGauge gauge, const void *in, const void *inNorm, 
		const int parity, const int dagger, const void *x, const void *xNorm, 
		const double k, const int volume, const int length, const QudaPrecision precision) {

  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (precision != gauge.precision)
    errorQuda("Mixing gauge and spinor precision not supported");

  if (precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    dslashCuda<2>((double2*)out, (float*)outNorm, (double2*)gauge0, (double2*)gauge1, 
		  gauge.reconstruct, (double2*)in, (float*)inNorm, parity, dagger, 
		  (double2*)x, (float*)xNorm, k, volume, length);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (precision == QUDA_SINGLE_PRECISION) {
    dslashCuda<4>((float4*)out, (float*)outNorm, (float4*)gauge0, (float4*)gauge1,
		  gauge.reconstruct, (float4*)in, (float*)inNorm, parity, dagger, 
		  (float4*)x, (float*)xNorm, k, volume, length);
  } else if (precision == QUDA_HALF_PRECISION) {
    dslashCuda<4>((short4*)out, (float*)outNorm, (short4*)gauge0, (short4*)gauge1,
		  gauge.reconstruct, (short4*)in, (float*)inNorm, parity, dagger, 
		  (short4*)x, (float*)xNorm, k, volume, length);
  }
  checkCudaError();

}


template <int N, typename spinorFloat, typename cloverFloat>
void cloverCuda(spinorFloat *out, float *outNorm, const cloverFloat *clover,
		const float *cloverNorm, const spinorFloat *in, const float *inNorm, 
		const int parity, const int volume, const int length)
{
  dim3 gridDim(volume/BLOCK_DIM, 1, 1);
  dim3 blockDim(BLOCK_DIM, 1, 1);

  int shared_bytes = blockDim.x*SHARED_FLOATS_PER_THREAD*bindSpinorTex<N>(length, in, inNorm);
  cloverKernel<<<gridDim, blockDim, shared_bytes>>> 
    (out, outNorm, clover, cloverNorm, in, inNorm, parity);
}

void cloverCuda(void *out, void *outNorm, const FullGauge gauge, const FullClover clover, 
		const void *in, const void *inNorm, const int parity, const int volume,
		const int length, const QudaPrecision precision) {

  void *cloverP, *cloverNormP;
  QudaPrecision clover_prec = bindCloverTex(clover, parity, &cloverP, &cloverNormP);

  if (precision != clover_prec)
    errorQuda("Mixing clover and spinor precision not supported");

  if (precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    cloverCuda<2>((double2*)out, (float*)outNorm, (double2*)cloverP, 
		  (float*)cloverNormP, (double2*)in, 
		  (float*)inNorm, parity, volume, length);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (precision == QUDA_SINGLE_PRECISION) {
    cloverCuda<4>((float4*)out, (float*)outNorm, (float4*)cloverP, 
		  (float*)cloverNormP, (float4*)in, 
		  (float*)inNorm, parity, volume, length);
  } else if (precision == QUDA_HALF_PRECISION) {
    cloverCuda<4>((short4*)out, (float*)outNorm, (short4*)cloverP, 
		  (float*)cloverNormP, (short4*)in,
		  (float*)inNorm, parity, volume, length);
  }
  checkCudaError();

}

// Clover wrappers
template <int N, typename spinorFloat, typename cloverFloat, typename gaugeFloat>
void cloverDslashCuda(spinorFloat *out, float *outNorm, const gaugeFloat gauge0, 
		      const gaugeFloat gauge1, const QudaReconstructType reconstruct, 
		      const cloverFloat *clover, const float *cloverNorm, const spinorFloat *in, 
		      const float* inNorm, const int parity, const int dagger, const spinorFloat *x, 
		      const float* xNorm, const double &a, const int volume, const int length)
{
  dim3 gridDim(volume/BLOCK_DIM, 1, 1);
  dim3 blockDim(BLOCK_DIM, 1, 1);

  int shared_bytes = blockDim.x*SHARED_FLOATS_PER_THREAD*bindSpinorTex<N>(length, in, inNorm, x, xNorm);

  if (x==0) { // not xpay
    if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	cloverDslash12Kernel <<<gridDim, blockDim, shared_bytes>>> 
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, parity);
      } else {
	cloverDslash12DaggerKernel <<<gridDim, blockDim, shared_bytes>>>
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, parity);
      }
    } else {
      if (!dagger) {
	cloverDslash8Kernel <<<gridDim, blockDim, shared_bytes>>> 	
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, parity);
      } else {
	cloverDslash8DaggerKernel <<<gridDim, blockDim, shared_bytes>>>
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, parity);
      }
    }
  } else { // doing xpay
    if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	cloverDslash12XpayKernel <<<gridDim, blockDim, shared_bytes>>> 
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, parity, x, xNorm, a);
      } else {
	cloverDslash12DaggerXpayKernel <<<gridDim, blockDim, shared_bytes>>>
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, parity, x, xNorm, a);
      }
    } else {
      if (!dagger) {
	cloverDslash8XpayKernel <<<gridDim, blockDim, shared_bytes>>> 	
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, parity, x, xNorm, a);
      } else {
	cloverDslash8DaggerXpayKernel <<<gridDim, blockDim, shared_bytes>>>
	  (out, outNorm, gauge0, gauge1, clover, cloverNorm, in, inNorm, parity, x, xNorm, a);
      }
    }
  }

}

void cloverDslashCuda(void *out, void *outNorm, const FullGauge gauge, const FullClover cloverInv,
		      const void *in, const void *inNorm, const int parity, const int dagger, 
		      const void *x, const void *xNorm, const double a, const int volume, 
		      const int length, const QudaPrecision precision) {

  void *cloverP, *cloverNormP;
  QudaPrecision clover_prec = bindCloverTex(cloverInv, parity, &cloverP, &cloverNormP);

  void *gauge0, *gauge1;
  bindGaugeTex(gauge, parity, &gauge0, &gauge1);

  if (precision != gauge.precision)
    errorQuda("Mixing gauge and spinor precision not supported");

  if (precision != clover_prec)
    errorQuda("Mixing clover and spinor precision not supported");

  if (precision == QUDA_DOUBLE_PRECISION) {
#if (__CUDA_ARCH__ >= 130)
    cloverDslashCuda<2>((double2*)out, (float*)outNorm, (double2*)gauge0, (double2*)gauge1, 
			gauge.reconstruct, (double2*)cloverP, (float*)cloverNormP, (double2*)in, 
			(float*)inNorm, parity, dagger, (double2*)x, (float*)xNorm, a, volume, length);
#else
    errorQuda("Double precision not supported on this GPU");
#endif
  } else if (precision == QUDA_SINGLE_PRECISION) {
    cloverDslashCuda<4>((float4*)out, (float*)outNorm, (float4*)gauge0, (float4*)gauge1, 
			gauge.reconstruct, (float4*)cloverP, (float*)cloverNormP, (float4*)in, 
			(float*)inNorm, parity, dagger, (float4*)x, (float*)xNorm, a, volume, length);
  } else if (precision == QUDA_HALF_PRECISION) {
    cloverDslashCuda<4>((short4*)out, (float*)outNorm, (short4*)gauge0, (short4*)gauge1, 
			gauge.reconstruct, (short4*)cloverP, (float*)cloverNormP, (short4*)in,
			(float*)inNorm, parity, dagger, (short4*)x, (float*)xNorm, a, volume, length);
  }

  checkCudaError();

}

#if 1
template <int spinorN, typename spinorFloat, typename fatGaugeFloat, typename longGaugeFloat>
  void staggeredDslashCuda(spinorFloat *out, float *outNorm, const fatGaugeFloat *fatGauge0, const fatGaugeFloat *fatGauge1, 
			   const longGaugeFloat* longGauge0, const longGaugeFloat* longGauge1, 
			   const QudaReconstructType reconstruct, const spinorFloat *in, const float *inNorm,
			   const int parity, const int dagger, const spinorFloat *x, const float *xNorm, 
			   const double &a, const int volume, const int length, const int ghost_length,
			   cudaColorSpinorField* inSpinor) {
    

  dim3 gridDim(volume/BLOCK_DIM, 1, 1);
  dim3 blockDim(BLOCK_DIM, 1, 1);

  int shared_bytes = blockDim.x*SHARED_FLOATS_PER_THREAD*bindSpinorTex_mg<spinorN>(length, ghost_length, in, inNorm, x, xNorm);

  hipEvent_t start1, stop1;
  hipEvent_t start2, stop2;
  hipEventCreate(&start1);
  hipEventCreate(&stop1);
  hipEventCreate(&start2);
  hipEventCreate(&stop2);
  
  struct timeval t0, t1;
  struct timeval t01, t02, t03;


  //hipDeviceSynchronize(); //sync whatever kernel that is not synced 

  gettimeofday(&t0, NULL);  	
  hipEventRecord(start1, stream[0]);

  if (x==0) { // not doing xpay
    if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	staggeredDslash12InternalKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1,
											 longGauge0, longGauge1, in, inNorm, parity); CUERR;
      } else {
	staggeredDslash12DaggerInternalKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
												longGauge0, longGauge1, in, inNorm, parity); CUERR;
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_8){
	  
      if (!dagger) {
	staggeredDslash8InternalKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
											 longGauge0, longGauge1, in, inNorm, parity); 
      } else {
	staggeredDslash8DaggerInternalKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1,
											      longGauge0, longGauge1, in, inNorm, parity); 
      }
    }else{
      errorQuda("Invalid reconstruct value(%d) in function %s\n", reconstruct, __FUNCTION__);
    }
  } else { // doing xpay
    
    if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	staggeredDslash12AxpyInternalKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
											      longGauge0, longGauge1, in, inNorm, parity, x, xNorm, a); CUERR;
      } else {
	staggeredDslash12DaggerAxpyInternalKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1,
												   longGauge0, longGauge1, in, inNorm, parity, x, xNorm, a); CUERR;
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_8) {
      if (!dagger) {
	staggeredDslash8AxpyInternalKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1,
											    longGauge0, longGauge1, in, inNorm, parity, x, xNorm, a);
      } else {
	staggeredDslash8DaggerAxpyInternalKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1, 
												  longGauge0, longGauge1, in, inNorm, parity, x, xNorm, a); 
      }
    }else{
      errorQuda("Invalid reconstruct value in function %s\n", __FUNCTION__);	  
    }    
  }


  exchange_gpu_spinor_start(inSpinor, fwd_nbr_spinor, back_nbr_spinor, f_norm, b_norm, &stream[1]);


  hipEventRecord(stop1, stream[0]);  
  gettimeofday(&t01, NULL);
  exchange_gpu_spinor_wait(inSpinor, fwd_nbr_spinor, back_nbr_spinor, f_norm, b_norm, &stream[1]);
  gettimeofday(&t02, NULL);    
  hipEventRecord(start2, stream[0]);

  hipStreamSynchronize(stream[0]);
  gettimeofday(&t03, NULL);    

  if (x==0) { // not doing xpay
    if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	staggeredDslash12BoundaryKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1, 
											 longGauge0, longGauge1, in, inNorm, parity); CUERR;
      } else {
	staggeredDslash12DaggerBoundaryKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>> (out, outNorm, fatGauge0, fatGauge1,
												longGauge0, longGauge1, in, inNorm, parity); CUERR;
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_8){
      
      if (!dagger) {
	staggeredDslash8BoundaryKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
											 longGauge0, longGauge1, in, inNorm, parity); CUERR;
      } else {
	staggeredDslash8DaggerBoundaryKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1, 
											      longGauge0, longGauge1, in, inNorm, parity); CUERR;
      }
    }else{
      errorQuda("Invalid reconstruct value(%d) in function %s\n", reconstruct, __FUNCTION__);
    }
  } else { // doing xpay
    
    if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	staggeredDslash12AxpyBoundaryKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
											      longGauge0, longGauge1, in, inNorm, parity, x, xNorm, a); CUERR;
      } else {
	staggeredDslash12DaggerAxpyBoundaryKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1,
												   longGauge0, longGauge1, in, inNorm, parity, x, xNorm, a); CUERR;
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_8) {
      if (!dagger) {
	staggeredDslash8AxpyBoundaryKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1,
											    longGauge0, longGauge1, in, inNorm, parity, x, xNorm, a); CUERR;
      } else {
	staggeredDslash8DaggerAxpyBoundaryKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1, 
												  longGauge0, longGauge1, in, inNorm, parity, x, xNorm, a); CUERR;
      }
    }else{
      errorQuda("Invalid reconstruct value in function %s\n", __FUNCTION__);	  
    }    
  }
  hipEventRecord(stop2, stream[0]);  


  hipDeviceSynchronize(); CUERR;  
  gettimeofday(&t1, NULL);
  
  float internal_time, boundary_time;
  hipEventElapsedTime(&internal_time, start1, stop1);
  hipEventElapsedTime(&boundary_time, start2, stop2);
  /*  
  PRINTF("Dlsash time is %.2f(ms),exchange_walltime=%.2f,  internal_kernel=%.2f(ms), boundary_kernel=%.2f(ms)\n",
	 TDIFF(t1, t0)*1000 , TDIFF(t02,t01)*1000, internal_time, boundary_time);
  PRINTF("CPU_internaltime=%.2f, CPU_boundarytime=%.2f\n", TDIFF(t01,t0)*1000, TDIFF(t1, t02)*1000);
  PRINTF("t03-t02=%.2f, pure boundary kernel=%.2f, walltime for internal kernel (t03-t1)=%.2f\n", 
	 TDIFF(t03, t02)*1000, TDIFF(t1, t03)*1000, TDIFF(t03, t0)*1000);
  */
}
#else
template <int spinorN, typename spinorFloat, typename fatGaugeFloat, typename longGaugeFloat>
  void staggeredDslashCuda(spinorFloat *out, float *outNorm, const fatGaugeFloat *fatGauge0, const fatGaugeFloat *fatGauge1, 
			   const longGaugeFloat* longGauge0, const longGaugeFloat* longGauge1, 
			   const QudaReconstructType reconstruct, const spinorFloat *in, const float *inNorm,
			   const int parity, const int dagger, const spinorFloat *x, const float *xNorm, 
			   const double &a, const int volume, const int length, const int ghost_length,
			   cudaColorSpinorField* inSpinor) {
    

  dim3 gridDim(volume/BLOCK_DIM, 1, 1);
  dim3 blockDim(BLOCK_DIM, 1, 1);

  int shared_bytes = blockDim.x*SHARED_FLOATS_PER_THREAD*bindSpinorTex_mg<spinorN>(length, ghost_length, in, inNorm, x, xNorm);

  //hipEvent_t start1, stop1;
  //hipEvent_t start2, stop2;
  //hipEventCreate(&start1);
  //hipEventCreate(&stop1);
  //hipEventCreate(&start2);
  //hipEventCreate(&stop2);
  
  struct timeval t0, t1, t2;
  //struct timeval t01, t02;
  
  hipDeviceSynchronize();
  
  gettimeofday(&t0, NULL);
  exchange_gpu_spinor(inSpinor, fwd_nbr_spinor, back_nbr_spinor, f_norm, b_norm, &stream[0]);
  hipDeviceSynchronize();
  gettimeofday(&t1, NULL);
  //hipEventRecord(start1, stream[0]);
  
  if (x==0) { // not doing xpay
    if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	staggeredDslash12Kernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1,
											 longGauge0, longGauge1, in, inNorm, parity); CUERR;
      } else {
	staggeredDslash12DaggerKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
												longGauge0, longGauge1, in, inNorm, parity); CUERR;
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_8){
      
      if (!dagger) {
	staggeredDslash8Kernel <<<gridDim, blockDim, shared_bytes, stream[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
											 longGauge0, longGauge1, in, inNorm, parity); CUERR;
      } else {
	staggeredDslash8DaggerKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1,
											      longGauge0, longGauge1, in, inNorm, parity); CUERR;
      }
    }else{
      errorQuda("Invalid reconstruct value(%d) in function %s\n", reconstruct, __FUNCTION__);
    }
  } else { // doing xpay
    
    if (reconstruct == QUDA_RECONSTRUCT_12) {
      if (!dagger) {
	staggeredDslash12AxpyKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
											      longGauge0, longGauge1, in, inNorm, parity, x, xNorm, a); CUERR;
      } else {
	staggeredDslash12DaggerAxpyKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1,
												   longGauge0, longGauge1, in, inNorm, parity, x, xNorm, a); CUERR;
      }
    } else if (reconstruct == QUDA_RECONSTRUCT_8) {
      if (!dagger) {
	staggeredDslash8AxpyKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1,
											    longGauge0, longGauge1, in, inNorm, parity, x, xNorm, a); CUERR;
      } else {
	staggeredDslash8DaggerAxpyKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1, 
												  longGauge0, longGauge1, in, inNorm, parity, x, xNorm, a); CUERR;
      }
    }else{
      errorQuda("Invalid reconstruct value in function %s\n", __FUNCTION__);	  
    }    
  }
  hipDeviceSynchronize();
  gettimeofday(&t2, NULL);  

  //hipEventRecord(stop1, stream[0]);  
  //gettimeofday(&t01, NULL);
  //gettimeofday(&t02, NULL);    
  //  hipEventRecord(start2, stream[0]);

 
  //hipEventRecord(stop2, stream[0]);  
  //hipDeviceSynchronize(); CUERR;
  //gettimeofday(&t1, NULL);
  
  //float internal_time, boundary_time;
  //hipEventElapsedTime(&internal_time, start1, stop1);
  //hipEventElapsedTime(&boundary_time, start2, stop2);
  
  PRINTF("exchange_walltime=%.2f,  kernel time=%.2f(ms)\n",
	 TDIFF(t1, t0)*1000 , TDIFF(t2,t1)*1000);
  
  
}

#endif


//This function is a special case for 18(no) reconstruct long link
//The reason is to make the type match easier(e.g float2 instead of float4)

template <int spinorN, typename spinorFloat, typename fatGaugeFloat, typename longGaugeFloat>
  void staggeredDslashNoReconCuda(spinorFloat *out, float *outNorm, const fatGaugeFloat *fatGauge0, const fatGaugeFloat *fatGauge1, 
				  const longGaugeFloat* longGauge0, const longGaugeFloat* longGauge1, 
				  const QudaReconstructType reconstruct, const spinorFloat *in, const float *inNorm,
				  const int parity, const int dagger, const spinorFloat *x, const float *xNorm, 
				  const double &a, const int volume, const int length, const int ghost_length,
				  cudaColorSpinorField* inSpinor) 
{
  
  
  dim3 gridDim(volume/BLOCK_DIM, 1, 1);
  dim3 blockDim(BLOCK_DIM, 1, 1);
  
  int shared_bytes = blockDim.x*SHARED_FLOATS_PER_THREAD*bindSpinorTex_mg<spinorN>(length, ghost_length, in, inNorm, x, xNorm);
  
  if (x==0) { // not doing xpay
    if (!dagger) {
      staggeredDslash18InternalKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1, 
										       longGauge0, longGauge1, in, inNorm, parity);CUERR;
    } else {
      staggeredDslash18DaggerInternalKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
											      longGauge0, longGauge1, in, inNorm, parity);CUERR;
    }    
  } else { // doing xpay
    
    if (!dagger) {
      staggeredDslash18AxpyInternalKernel<<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1,
											  longGauge0, longGauge1, in, inNorm, parity, x, xNorm, a); CUERR;
    } else {
      staggeredDslash18DaggerAxpyInternalKernel<<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1,
												longGauge0, longGauge1, in, inNorm, parity, x, xNorm, a); CUERR;
    }          
  }
  exchange_gpu_spinor_start(inSpinor, fwd_nbr_spinor, back_nbr_spinor, f_norm, b_norm, &stream[1]);   
  exchange_gpu_spinor_wait(inSpinor, fwd_nbr_spinor, back_nbr_spinor, f_norm, b_norm, &stream[1]); 
  
  if (x==0) { // not doing xpay
    if (!dagger) {
      staggeredDslash18BoundaryKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1, 
										       longGauge0, longGauge1, in, inNorm, parity);CUERR;
    } else {
      staggeredDslash18DaggerBoundaryKernel <<<gridDim, blockDim, shared_bytes, stream[0]>>> (out, outNorm, fatGauge0, fatGauge1, 
											      longGauge0, longGauge1, in, inNorm, parity);CUERR;
    }    
  } else { // doing xpay
    
    if (!dagger) {
      staggeredDslash18AxpyBoundaryKernel<<<gridDim, blockDim, shared_bytes, stream[0]>>>	(out, outNorm, fatGauge0, fatGauge1,
												 longGauge0, longGauge1, in, inNorm, parity, x, xNorm, a); CUERR;
    } else {
      staggeredDslash18DaggerAxpyBoundaryKernel<<<gridDim, blockDim, shared_bytes, stream[0]>>>(out, outNorm, fatGauge0, fatGauge1,
												longGauge0, longGauge1, in, inNorm, parity, x, xNorm, a); CUERR;
    }          
  }
  
  hipDeviceSynchronize();
}

#if (__CUDA_ARCH__ >= 200)
__global__ void dummykernel()
{
  
}

#endif
void staggeredDslashCuda(void *out, void *outNorm, const FullGauge fatGauge, const FullGauge longGauge, 
			 cudaColorSpinorField *in,
			 const int parity, const int dagger, const void *x, const void *xNorm, 
			 const double k, const int volume, const int length, const int ghost_length, const QudaPrecision precision) 
{

#if (__CUDA_ARCH__ >= 200)
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(dummykernel), hipFuncCachePreferL1);
  dummykernel<<<1,1>>>();
#endif
  
  for(int i=0;i < 2 ;i ++){
    hipStreamCreate(&stream[i]); CUERR;
  }

  void *fatGauge0, *fatGauge1;
  void* longGauge0, *longGauge1;
  bindFatGaugeTex(fatGauge, parity, &fatGauge0, &fatGauge1);
  bindLongGaugeTex(longGauge, parity, &longGauge0, &longGauge1);
    
  if (precision != fatGauge.precision || precision != longGauge.precision){
    errorQuda("Mixing gauge and spinor precision not supported");
  }
    


  if (precision == QUDA_DOUBLE_PRECISION) {
    if (longGauge.reconstruct == QUDA_RECONSTRUCT_NO){
      staggeredDslashNoReconCuda<2>((double2*)out, (float*)outNorm, (double2*)fatGauge0, (double2*)fatGauge1, 			       
				    (double2*)longGauge0, (double2*)longGauge1,
				    longGauge.reconstruct, (double2*)in->getV(), (float*)in->getNorm(), parity, dagger, 
				    (double2*)x, (float*)xNorm, k, volume, length, ghost_length, in);
    }else{
      staggeredDslashCuda<2>((double2*)out, (float*)outNorm, (double2*)fatGauge0, (double2*)fatGauge1, 			       
			     (double2*)longGauge0, (double2*)longGauge1,
			     longGauge.reconstruct, (double2*)in->getV(), (float*)in->getNorm(), parity, dagger, 
			     (double2*)x, (float*)xNorm, k, volume, length, ghost_length, in);
    }
    
  } else if (precision == QUDA_SINGLE_PRECISION) {
    if (longGauge.reconstruct == QUDA_RECONSTRUCT_NO){
      staggeredDslashNoReconCuda<2>((float2*)out, (float*)outNorm, (float2*)fatGauge0, (float2*)fatGauge1,
				    (float2*)longGauge0, (float2*)longGauge1,
				    longGauge.reconstruct, (float2*)in->getV(), (float*)in->getNorm(), parity, dagger, 
				    (float2*)x, (float*)xNorm, k, volume, length, ghost_length, in);
    }else{
      staggeredDslashCuda<2>((float2*)out, (float*)outNorm, (float2*)fatGauge0, (float2*)fatGauge1,
			     (float4*)longGauge0, (float4*)longGauge1,
			     longGauge.reconstruct, (float2*)in->getV(), (float*)in->getNorm(), parity, dagger, 
			     (float2*)x, (float*)xNorm, k, volume, length, ghost_length, in);
    }
  } else if (precision == QUDA_HALF_PRECISION) {	
    if (longGauge.reconstruct == QUDA_RECONSTRUCT_NO){
      staggeredDslashNoReconCuda<2>((short2*)out, (float*)outNorm, (short2*)fatGauge0, (short2*)fatGauge1,
				    (short2*)longGauge0, (short2*)longGauge1,
				    longGauge.reconstruct, (short2*)in->getV(), (float*)in->getNorm(), parity, dagger, 
				    (short2*)x, (float*)xNorm, k, volume, length, ghost_length, in);
    }else{
      staggeredDslashCuda<2>((short2*)out, (float*)outNorm, (short2*)fatGauge0, (short2*)fatGauge1,
			     (short4*)longGauge0, (short4*)longGauge1,
			     longGauge.reconstruct, (short2*)in->getV(), (float*)in->getNorm(), parity, dagger, 
			     (short2*)x, (float*)xNorm, k, volume, length, ghost_length, in);
    }
  }

  for (int i = 0; i < 2; i++) {
    hipStreamDestroy(stream[i]);
  }
  
}


